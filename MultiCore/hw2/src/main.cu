#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "config.h"

//**************************************************************************
#include <cmath>
#include <iostream>
using namespace std;
__constant__ float const_log[26];
texture<float, 1> texture_log;

// Baseline
__global__ void v0_baseline(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k]) 
                ans -= (float)cnt[k] *  (1.0 / count) * log2f((float)cnt[k]/count);
        out[idy*width+idx] = ans;
    }
}

// 计数器cnt用unsigned char代替int
__global__ void v1_char(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * log2f((float)cnt[k]/count);
        out[idy*width+idx] = ans;
    }
}

// 预处理对数表至寄存器
__global__ void v2_registerTable(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char cnt[16] = {0}, count = 0;
    const float pre_log[26] = { 0.0, log2f(1.0), log2f(2.0), log2f(3.0), log2f(4.0), log2f(5.0),
                                log2f(6.0), log2f(7.0), log2f(8.0), log2f(9.0), log2f(10.0),
                                log2f(11.0), log2f(12.0), log2f(13.0), log2f(14.0), log2f(15.0),
                                log2f(16.0), log2f(17.0), log2f(18.0), log2f(19.0), log2f(20.0),
                                log2f(21.0), log2f(22.0), log2f(23.0), log2f(24.0), log2f(25.0)};
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * (pre_log[cnt[k]]-pre_log[count]);
        out[idy*width+idx] = ans;
    }
}

// 预处理对数表至全局内存
__global__ void v3_globalTable(int *in, float *out, int height, int width, float *global_log) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * (global_log[cnt[k]]-global_log[count]);
        out[idy*width+idx] = ans;
    }
}

// 预处理对数表至纹理内存
__global__ void v4_textureTable(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * (tex1Dfetch(texture_log, cnt[k])-tex1Dfetch(texture_log, count));
        out[idy*width+idx] = ans;
    }
}

// 预处理对数表至常量内存
__global__ void v5_constTable(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * (const_log[cnt[k]]-const_log[count]);
        out[idy*width+idx] = ans;
    }
}

// 预处理对数表至共享内存
__global__ void v6_sharedTable(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    __shared__ float shared_log[26];
    if (tid != 0 && tid < 26)
        shared_log[tid] = log2f((float)tid);
    __syncthreads();
    unsigned char cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * (shared_log[cnt[k]]-shared_log[count]);
        out[idy*width+idx] = ans;
    }
}

// 最佳优化版本
__global__ void v7_optimal(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    extern __shared__ unsigned char cnt[][16];
    memset(cnt + tid, 0, 16 * sizeof(unsigned char));
    __syncthreads();
    unsigned char count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[tid][in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k) 
            ans -= cnt[tid][k] ? cnt[tid][k] * (1.0 / count) * (const_log[cnt[tid][k]]-const_log[count]) : 0;
        out[idy*width+idx] = ans;
    }
}
//**************************************************************************

int main(int argc, char* argv[]) {
    if (argc == 3) {
        inputPath = argv[1];
        outputPath = argv[2];
    }

    // Open the input file
    FILE *stream = fopen(inputPath, "rb");
    if (stream == NULL) {
        printf("failed to open the data file %s\n", inputPath);
        return -1;
    }

    // Open a stream to write out results in text
    FILE *outStream = fopen(outputPath, "wb");
    if (outStream == NULL) {
        printf("failed to open the output file %s\n", outputPath);
        return -1;
    }

    // Read in and process the input matrix one-by-one
    int width, height, size;
    int *input;
    float *result;
    loadMatrix(stream, &width, &height, &input);
    size = width * height;
    result = (float*)malloc(sizeof(float) * size);

    //**************************************************************************
    int *d_in;
    float *d_out, *d_log;
    // 预处理
    float pre_log[26] = { 0.0, log2f(1.0), log2f(2.0), log2f(3.0), log2f(4.0), log2f(5.0),
                        log2f(6.0), log2f(7.0), log2f(8.0), log2f(9.0), log2f(10.0),
                        log2f(11.0), log2f(12.0), log2f(13.0), log2f(14.0), log2f(15.0),
                        log2f(16.0), log2f(17.0), log2f(18.0), log2f(19.0), log2f(20.0),
                        log2f(21.0), log2f(22.0), log2f(23.0), log2f(24.0), log2f(25.0)};
    // 显存分配
    CHECK(hipMalloc((void**)&d_in, size * sizeof(int)));
    CHECK(hipMalloc((void**)&d_out, size * sizeof(float)));
    CHECK(hipMalloc((void**)&d_log, sizeof(pre_log)));
    // 数据拷贝
    CHECK(hipMemcpy((void*)d_in, (void*)input, size * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy((void*)d_log, (void*)pre_log, sizeof(pre_log), hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_log), (const float*)pre_log, sizeof(pre_log)));
    CHECK(hipBindTexture(0, texture_log, d_log, sizeof(pre_log)));
    // blockSize和gridSize
    dim3 blockSize(8, 8);
    dim3 gridSize(divup(width, blockSize.x), divup(height, blockSize.y));
    // 记录时间
    long st, et;
    st = getTime();
    // 调用核函数
    int v = 7;
    switch (v) {
        case 0:
            printf("Baseline: \n");
            v0_baseline <<<gridSize, blockSize>>> (d_in, d_out, height, width);
            break;
        case 1:
            printf("Unsigned Char\n");
            v1_char <<<gridSize, blockSize>>> (d_in, d_out, height, width);
            break;
        case 2:
            printf("Log Table in Register: \n");
            v2_registerTable <<<gridSize, blockSize>>> (d_in, d_out, height, width);
            break;
        case 3:
            printf("Log Table in Global Memory: \n");
            v3_globalTable <<<gridSize, blockSize>>> (d_in, d_out, height, width, d_log);
            break;
        case 4:
            printf("Log Table in Texture Memory: \n");
            v4_textureTable <<<gridSize, blockSize>>> (d_in, d_out, height, width);
            break;
        case 5:
            printf("Log Table in Constant Memory: \n");
            v5_constTable <<<gridSize, blockSize>>> (d_in, d_out, height, width);
            break;
        case 6:
            printf("Log Table in Shared Memory: \n");
            v6_sharedTable <<<gridSize, blockSize>> >(d_in, d_out, height, width);
            break;
        case 7:
            printf("Optimal: \n");
            v7_optimal <<<gridSize, blockSize, 16 * 16 * 16 * sizeof(unsigned char)>>> (d_in, d_out, height, width);
            break;
        default:
            break;
    }
    // 输出运行时间
    et = getTime();
    printf("Time cost (CUDA): %.3f ms\n", et-st);
    // 拷贝回host并释放显存
    CHECK(hipUnbindTexture(texture_log));
    CHECK(hipMemcpy((void*)result, (void*)d_out, size * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_out));
    CHECK(hipFree(d_log));
    //**************************************************************************
       
    saveMatrix(outStream, &width, &height, &result);

    // De-allocate the input and the result
    free(input);
    input = NULL;
    free(result);
    result = NULL;
    
    // Close the stream
    fclose(stream);
    fclose(outStream);
    return 0;
}