#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include "utils.h"

__constant__ float const_log[26];
texture<float, 1> texture_log;

// Baseline
__global__ void v0_baseline(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k]) 
                ans -= (float)cnt[k] *  (1.0 / count) * log2f((float)cnt[k]/count);
        out[idy*width+idx] = ans;
    }
}

// 计数器cnt用unsigned char代替int
__global__ void v1_char(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * log2f((float)cnt[k]/count);
        out[idy*width+idx] = ans;
    }
}

// 预处理对数表至寄存器
__global__ void v2_registerTable(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char cnt[16] = {0}, count = 0;
    const float pre_log[26] = { 0.0, log2f(1.0), log2f(2.0), log2f(3.0), log2f(4.0), log2f(5.0),
                                log2f(6.0), log2f(7.0), log2f(8.0), log2f(9.0), log2f(10.0),
                                log2f(11.0), log2f(12.0), log2f(13.0), log2f(14.0), log2f(15.0),
                                log2f(16.0), log2f(17.0), log2f(18.0), log2f(19.0), log2f(20.0),
                                log2f(21.0), log2f(22.0), log2f(23.0), log2f(24.0), log2f(25.0)};
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * (pre_log[cnt[k]]-pre_log[count]);
        out[idy*width+idx] = ans;
    }
}

// 预处理对数表至全局内存
__global__ void v3_globalTable(int *in, float *out, int height, int width, float *global_log) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * (global_log[cnt[k]]-global_log[count]);
        out[idy*width+idx] = ans;
    }
}

// 预处理对数表至纹理内存
__global__ void v4_textureTable(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * (tex1Dfetch(texture_log, cnt[k])-tex1Dfetch(texture_log, count));
        out[idy*width+idx] = ans;
    }
}

// 预处理对数表至常量内存
__global__ void v5_constTable(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * (const_log[cnt[k]]-const_log[count]);
        out[idy*width+idx] = ans;
    }
}

// 预处理对数表至共享内存
__global__ void v6_sharedTable(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    __shared__ float shared_log[26];
    if (tid != 0 && tid < 26)
        shared_log[tid] = log2f((float)tid);
    __syncthreads();
    unsigned char cnt[16] = {0}, count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k)
            if (cnt[k])
                ans -= (float)cnt[k] *  (1.0 / count) * (shared_log[cnt[k]]-shared_log[count]);
        out[idy*width+idx] = ans;
    }
}

// 最佳优化版本
__global__ void v7_optimal(int *in, float *out, int height, int width) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    extern __shared__ unsigned char cnt[][16];
    memset(cnt + tid, 0, 16 * sizeof(unsigned char));
    __syncthreads();
    unsigned char count = 0;
    if (idy < height && idx < width) {
        for (int dy = -2; dy <= 2; ++dy)
            for (int dx = -2; dx <= 2; ++dx) {
                const int x = idx + dx, y = idy + dy;
                if (y >= 0 && y < height && x >= 0 && x < width) {
                    ++cnt[tid][in[y*width+x]];
                    ++count;
                }
            }
        float ans = 0;
        for (int k = 0; k < 16; ++k) 
            ans -= cnt[tid][k] ? cnt[tid][k] * (1.0 / count) * (const_log[cnt[tid][k]]-const_log[count]) : 0;
        out[idy*width+idx] = ans;
    }
}


