#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <cmath>
#include <cstring>
#include "kernel.cu"
#include "utils.h"
using namespace std;

int main(int argc, char *argv[]) {
    int *in, *d_in;
    float *out, *d_out, *d_log;
    // 随机初始化矩阵
    int m = strtol(argv[1], NULL, 10);
    int n = strtol(argv[2], NULL, 10);
    Generate(&in, &out, m, n);
    // 线程块大小
    int b1 = strtol(argv[3], NULL, 10);
    int b2 = strtol(argv[4], NULL, 10);
    // CUDA优化版本号0-7
    int v = strtol(argv[5], NULL, 10);
    // 预处理
    float pre_log[26] = { 0.0, log2f(1.0), log2f(2.0), log2f(3.0), log2f(4.0), log2f(5.0),
                        log2f(6.0), log2f(7.0), log2f(8.0), log2f(9.0), log2f(10.0),
                        log2f(11.0), log2f(12.0), log2f(13.0), log2f(14.0), log2f(15.0),
                        log2f(16.0), log2f(17.0), log2f(18.0), log2f(19.0), log2f(20.0),
                        log2f(21.0), log2f(22.0), log2f(23.0), log2f(24.0), log2f(25.0)};
    // 显存分配
    CHECK(hipMalloc((void**)&d_in, m * n * sizeof(int)));
    CHECK(hipMalloc((void**)&d_out, m * n * sizeof(float)));
    CHECK(hipMalloc((void**)&d_log, sizeof(pre_log)));
    // 数据拷贝
    CHECK(hipMemcpy((void*)d_in, (void*)in, m * n * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy((void*)d_log, (void*)pre_log, sizeof(pre_log), hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_log), (const float*)pre_log, sizeof(pre_log)));
    CHECK(hipBindTexture(0, texture_log, d_log, sizeof(pre_log)));
    // blockSize和gridSize
    dim3 blockSize(b1, b2);
    dim3 gridSize(ceil((float) n / blockSize.x),
                  ceil((float) m / blockSize.y));
    // 记录时间
    hipEvent_t t1, t2;
    hipEventCreate(&t1);
    hipEventCreate(&t2);
    // 调用核函数
    switch (v) {
        case 0:
            printf("Baseline: \n");
            hipEventRecord(t1, 0);
            v0_baseline <<<gridSize, blockSize>>> (d_in, d_out, m, n);
            hipEventRecord(t2, 0);
            break;
        case 1:
            printf("Unsigned Char\n");
            hipEventRecord(t1, 0);
            v1_char <<<gridSize, blockSize>>> (d_in, d_out, m, n);
            hipEventRecord(t2, 0);
            break;
        case 2:
            printf("Log Table in Register: \n");
            hipEventRecord(t1, 0);
            v2_registerTable <<<gridSize, blockSize>>> (d_in, d_out, m, n);
            hipEventRecord(t2, 0);
            break;
        case 3:
            printf("Log Table in Global Memory: \n");
            hipEventRecord(t1, 0);
            v3_globalTable <<<gridSize, blockSize>>> (d_in, d_out, m, n, d_log);
            hipEventRecord(t2, 0);
            break;
        case 4:
            printf("Log Table in Texture Memory: \n");
            hipEventRecord(t1, 0);
            v4_textureTable <<<gridSize, blockSize>>> (d_in, d_out, m, n);
            hipEventRecord(t2, 0);
            break;
        case 5:
            printf("Log Table in Constant Memory: \n");
            hipEventRecord(t1, 0);
            v5_constTable <<<gridSize, blockSize>>> (d_in, d_out, m, n);
            hipEventRecord(t2, 0);
            break;
        case 6:
            printf("Log Table in Shared Memory: \n");
            hipEventRecord(t1, 0);
            v6_sharedTable <<<gridSize, blockSize>> >(d_in, d_out, m, n);
            hipEventRecord(t2, 0);
            break;
        case 7:
            printf("Optimal: \n");
            hipEventRecord(t1, 0);
            v7_optimal <<<gridSize, blockSize, b1 * b2 * 16 * sizeof(unsigned char)>>> (d_in, d_out, m, n);
            hipEventRecord(t2, 0);
            break;
        default:
            hipEventRecord(t1, 0);
            hipEventRecord(t2, 0);
            break;
    }
    // 输出运行时间
    hipEventSynchronize(t1);
    hipEventSynchronize(t2);
    float t;
    hipEventElapsedTime(&t, t1, t2);
    printf("Time cost (CUDA): %.3f ms\n", t);
    hipEventDestroy(t1);
    hipEventDestroy(t2);
    // 拷贝回host并释放显存
    CHECK(hipUnbindTexture(texture_log));
    CHECK(hipMemcpy((void*)out, (void*)d_out, m * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_out));
    CHECK(hipFree(d_log));
    // 检验正确性
    Evaluate(in, out, m, n);
    return 0;
}