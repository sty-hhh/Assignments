
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cmath>
using namespace std;

// 随机初始化两个 m*n 大小的矩阵
void Generate(float **a, float **b, float **c, int m, int n) {
    *a = new float[m*n], *b = new float [m*n], *c = new float [m*n];
    for (int i = 0; i < m; ++i) 
        for (int j = 0; j < n; ++j) {
            (*a)[i*n+j] = 10.0 * rand()/(RAND_MAX+1.0);
            (*b)[i*n+j] = 10.0 * rand()/(RAND_MAX+1.0);
        }
}
// 检验矩阵加法计算结果
void Evaluate(float *a, float *b, float *c, int m, int n) {
    for (int i = 0; i < m; ++i) 
        for (int j = 0; j < n; ++j) 
            if ((fabs(a[i*n+j] + b[i*n+j]- c[i*n+j]) / c[i*n+j]) > 1e-4) {
                printf("Computation Error In %d Row %d Col!\n", i, j);
                return;
            }
    printf("Computation Correct!\n");
}
// 一维加法
__global__ void MatrixMul_1d(float *a, float *b, float *c, int m, int n, int sz) {
    int id = (threadIdx.x + blockIdx.x * blockDim.x) * sz;
    for (int i = id; i < id + sz; ++i)
        if (i < m * n)  
            c[i] = a[i] + b[i];
}
// 二维加法
__global__ void MatrixMul_2d(float *a, float *b, float *c, int m, int n, int sz) {
    int idx = (threadIdx.x + blockIdx.x * blockDim.x) * sz;
    int idy = (threadIdx.y + blockIdx.y * blockDim.y) * sz;
    for (int y = idy; y < idy + sz; ++y) 
        for (int x = idx; x < idx + sz; ++x) 
            if (y < m && x < n) {
                int id = y * n + x;
                c[id] = a[id] + b[id];
            }    
}

int main(int argc, char *argv[]) {
    float *a, *b, *c, *da, *db, *dc, t;
    int m = strtol(argv[1], NULL, 10);
    int n = strtol(argv[2], NULL, 10);
    // 随机初始化矩阵
    Generate(&a, &b, &c, m, n);
    // 显存分配
    hipMalloc((void**)&da, m*n*sizeof(float));
    hipMalloc((void**)&db, m*n*sizeof(float));
    hipMalloc((void**)&dc, m*n*sizeof(float));
    // 数据拷贝
    hipMemcpy((void*)da, (void*)a, m*n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)db, (void*)b, m*n*sizeof(float), hipMemcpyHostToDevice);
    // 设置blockSize和gridSize
    int b1 = strtol(argv[3], NULL, 10);         // blockSize一维
    int b2 = strtol(argv[4], NULL, 10);         // blockSize二维（1代表一维块）
    int grid_dim = strtol(argv[5], NULL, 10);   // grid维度
    int sz = strtol(argv[6], NULL, 10);         
    if (grid_dim == 1)  
        sz = sz * sz;
    dim3 blockSize(b1, b2);
    dim3 gridSize;
    // block一维, grid一维
    if (b2 == 1 && grid_dim == 1)  
        gridSize = dim3(ceil(ceil((float)m*n/sz) / blockSize.x));
    // block一维，grid二维 或 block二维，grid二维
    else if (grid_dim == 2)  
        gridSize = dim3(ceil(ceil((float)n/sz) / blockSize.x), ceil(ceil((float)m/sz) / blockSize.y));
    // 记录时间
    hipEvent_t t1, t2;
    hipEventCreate(&t1);
    hipEventCreate(&t2);
    hipEventRecord(t1, 0);
    // 调用核函数
    if (grid_dim == 1)  
        MatrixMul_1d <<<gridSize, blockSize>>> (da, db, dc, m, n, sz);
    else    
        MatrixMul_2d <<<gridSize, blockSize>>> (da, db, dc, m, n, sz);
    // 输出运行时间
    hipEventRecord(t2, 0);
    hipEventSynchronize(t1);
    hipEventSynchronize(t2);
    hipEventElapsedTime(&t, t1, t2);
    printf("Time cost(CUDA): %.3f ms\n", t);
    hipEventDestroy(t1);
    hipEventDestroy(t2);
    // 拷贝回host并释放显存
    hipMemcpy((void*)c, (void*)dc, m*n*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    // 检验正确性
    Evaluate(a, b, c, m, n);
    return 0;
}
