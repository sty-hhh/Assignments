#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "config.h"

//**************************************************************************
// 一维加法
__global__ void MatrixMul_1d(float *a, float *b, float *c, int m, int n, int sz) {
    int id = (threadIdx.x + blockIdx.x * blockDim.x) * sz;
    for (int i = id; i < id + sz; ++i)
        if (i < m * n)  
            c[i] = a[i] + b[i];
}
// 二维加法
__global__ void MatrixMul_2d(float *a, float *b, float *c, int m, int n, int sz) {
    int idx = (threadIdx.x + blockIdx.x * blockDim.x) * sz;
    int idy = (threadIdx.y + blockIdx.y * blockDim.y) * sz;
    for (int y = idy; y < idy + sz; ++y) 
        for (int x = idx; x < idx + sz; ++x) 
            if (y < m && x < n) {
                int id = y * n + x;
                c[id] = a[id] + b[id];
            }    
}
//**************************************************************************

int main(int argc, char* argv[]) {
    if (argc == 3) {
        inputPath = argv[1];
        outputPath = argv[2];
    }

    // Open the input file
    FILE *stream = fopen(inputPath, "rb");
    if (stream == NULL) {
        printf("failed to open the data file %s\n", inputPath);
        return -1;
    }

    // Open a stream to write out results in text
    FILE *outStream = fopen(outputPath, "wb");
    if (outStream == NULL) {
        printf("failed to open the output file %s\n", outputPath);
        return -1;
    }

    // Read in and process the input matrix one-by-one
    int width, height, size;
    float *input1, *input2, *result;
    loadMatrix(stream, &width, &height, &input1);
    loadMatrix(stream, &width, &height, &input2);
    size = width * height;
    result = (float*)malloc(sizeof(float) * size);

    //**************************************************************************
    // 显存分配
    float *da, *db, *dc;
    CHECK(hipMalloc((void**)&da, size*sizeof(float)));
    CHECK(hipMalloc((void**)&db, size*sizeof(float)));
    CHECK(hipMalloc((void**)&dc, size*sizeof(float)));
    // 数据拷贝
    CHECK(hipMemcpy((void*)da, (void*)input1, size*sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy((void*)db, (void*)input2, size*sizeof(float), hipMemcpyHostToDevice));
    // 设置blockSize和gridSize
    int b1 = 32;         // blockSize一维
    int b2 = 32;         // blockSize二维（1代表一维块）
    int grid_dim = 2;    // grid维度
    int sz = 2;          // 每个线程计算的元素数（平方根）
    if (grid_dim == 1)  
        sz = sz * sz;
    dim3 blockSize(b1, b2);
    dim3 gridSize;
    // block一维, grid一维
    if (b2 == 1 && grid_dim == 1)  
        gridSize = dim3(divup(divup(size, sz), blockSize.x));
    // block一维，grid二维 或 block二维，grid二维
    else if (grid_dim == 2)  
        gridSize = dim3(divup(divup(width, sz), blockSize.x), divup(divup(height, sz), blockSize.y));

    long st, et;
    st = getTime();
    // 调用核函数
    if (grid_dim == 1)  
        MatrixMul_1d <<<gridSize, blockSize>>> (da, db, dc, height, width, sz);
    else    
        MatrixMul_2d <<<gridSize, blockSize>>> (da, db, dc, height, width, sz);
    et = getTime();
    printf("Time cost (CUDA): %.3f ms\n", (et - st) / 1e6);

    // 拷贝回host并释放显存
    CHECK(hipMemcpy((void*)result, (void*)dc, size*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(da));
    CHECK(hipFree(db));
    CHECK(hipFree(dc));
    //**************************************************************************
       
    saveMatrix(outStream, &width, &height, &result);

    // De-allocate the nput and the result
    free(input1);
    free(input2);
    input1 = input2 = NULL;
    free(result);
    result = NULL;
    

    // Close the output stream
    fclose(outStream);
    return 0;
}